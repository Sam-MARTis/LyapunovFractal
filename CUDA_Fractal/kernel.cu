#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <iostream>
#define SEQLEN_MAX 100
#define TX 32
#define TY 16
#define seed 42

int divUp(int a, int b)
{
    return (a - 1 + b) / b;
}

__device__ unsigned char clip(int n)
{
    return n > 255 ? 255 : (n < 0 ? 0 : n);
}

__global__ void lyapunovCalcKernel(uchar4 *d_out, bool *seq, const int seqLen, float dx, float dy, int n, int m, float xStart, float yStart, float width, float height, int numIterations)
{
    const int xIdx = threadIdx.x + blockDim.x * blockIdx.x;
    const int yIdx = threadIdx.y + blockDim.y * blockIdx.y;
    if ((xIdx >= n) || (yIdx >= m))
        return;
    const int idx = xIdx + ((n - 1) - yIdx) * n;

    const int SL = seqLen;
    const float B = ((float)xIdx) * dx + xStart;
    const float A = ((float)yIdx) * dy + yStart;
    float seqArr[SEQLEN_MAX];
    for (int i = 0; i < SL; i++)
    {
        if (seq[i])
        {
            seqArr[i] = A;
        }
        else
        {
            seqArr[i] = B;
        }
    }

    hiprandState state;
    hiprand_init(seed, xIdx, yIdx, &state);
    float x = hiprand_uniform(&state) * 0.9999; // To handle edge case when hiprand returns 1
    if (x == 0.5)
    {
        x = 0.4999;
        printf("x is 0.5\n");
    }

    float sum = x;
    int j = 0;
    for (int i = 0; i < numIterations; i++)
    {

        x = seqArr[j] * x * (1 - x);
        sum += log(abs(seqArr[j] * (1 - 2 * x)));
        if (sum == INFINITY)
        {
            return;
        }
        j = (j + 1) % SL;
    }

    const float lyapunovExponent = sum / ((float)numIterations);

    if (lyapunovExponent < 0)
    {
        const int intensity = clip(round(255 * (-lyapunovExponent)));
        d_out[idx].x = clip(255 - intensity);
        d_out[idx].y = clip(255 - intensity);
        d_out[idx].z = 10;
    }
    else
    {
        const int intensity = clip(round(255 * (lyapunovExponent)));
        d_out[idx].x = 10;
        d_out[idx].y = 10;
        d_out[idx].z = clip(255 - intensity);
    }

    d_out[idx].w = 255;
}

void lyapunovKernelLauncher(uchar4 *out, bool *sequence, int sequenceLength, float xStart, float yStart, float width, float height, int n, int m, int numIterations)
{
    uchar4 *d_out = 0;
    bool *d_seq = 0;
    hipMalloc(&d_seq, sequenceLength * sizeof(bool));
    hipMemcpy(d_seq, sequence, sequenceLength * sizeof(bool), hipMemcpyHostToDevice);
    hipMalloc(&d_out, n * m * sizeof(uchar4));
    float dx = (width) / ((float)n);
    float dy = (height) / ((float)m);
    const dim3 blockSize(TX, TY);
    const dim3 gridSize(divUp(n, TX), divUp(m, TY));

    lyapunovCalcKernel<<<gridSize, blockSize>>>(d_out, d_seq, sequenceLength, dx, dy, n, m, xStart, yStart, width, height, numIterations);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << "\n";
    }
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, n * m * sizeof(uchar4), hipMemcpyDeviceToHost);
    hipFree(d_out);
    hipFree(d_seq);
}
